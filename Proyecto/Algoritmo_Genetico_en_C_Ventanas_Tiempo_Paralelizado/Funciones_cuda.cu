#include "hip/hip_runtime.h"
#include "Biblioteca_cuda.h"
#include <math.h> // para log2, etc.

// ----------------------------------------------------
// Implementación de la macro para manejo de errores
// ----------------------------------------------------
// Definición de la función
void gpuAssert(hipError_t code, const char *file, int line, bool abort) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// ----------------------------------------------------
// Kernels y funciones CUDA
// ----------------------------------------------------
__global__ void setup_curand_kernel(hiprandState *states, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

void obtenerConfiguracionCUDA(int *blockSize, int *minGridSize, int *gridSize, int N) {
    hipOccupancyMaxPotentialBlockSize(minGridSize, blockSize, evaluar_poblacion_kernel, 0, N);
    *gridSize = (N + *blockSize - 1) / *blockSize;
}

__global__ void evaluar_poblacion_kernel(individuo_gpu *poblacion, double *distancias, double *ventanas_tiempo,
                                         int tamano_poblacion, int longitud_genotipo) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tamano_poblacion) return;

    double total_cost = 0.0;      // Costo total del recorrido (en horas)
    double tiempo_acumulado = 0.0; // Tiempo transcurrido desde el inicio del recorrido

    int *genotipo = poblacion[idx].genotipo;

    // Iteramos sobre las ciudades en el genotipo de forma circular
    for (int i = 0; i < longitud_genotipo; i++) {
        int origen = genotipo[i];
        int destino = genotipo[(i + 1) % longitud_genotipo];

        // Añadimos el tiempo de viaje entre ciudades
        double tiempo_viaje = distancias[origen * longitud_genotipo + destino];
        tiempo_acumulado += tiempo_viaje;

        // Calculamos la hora de llegada al destino (ajustado al formato de 24 horas)
        double hora_llegada = fmod(tiempo_acumulado, 24.0);

        // Ventanas de tiempo de la ciudad de destino
        double ventana_inicio = ventanas_tiempo[destino * 2];
        double ventana_fin = ventanas_tiempo[destino * 2 + 1];

        // Ajustamos el tiempo acumulado si se llega fuera de la ventana permitida
        if (hora_llegada < ventana_inicio) {
            // Esperamos hasta el inicio de la ventana
            tiempo_acumulado += (ventana_inicio - hora_llegada);
        } else if (ventana_fin < ventana_inicio) {
            // Caso especial: La ventana cruza medianoche (ej. 22:00 a 02:00)
            if (hora_llegada > ventana_fin && hora_llegada < ventana_inicio) {
                tiempo_acumulado += (24.0 - hora_llegada + ventana_inicio); // Esperamos al siguiente día
            }
        } else if (hora_llegada > ventana_fin) {
            // Esperamos al siguiente día si llegamos después del cierre
            tiempo_acumulado += (24.0 - hora_llegada + ventana_inicio);
        }

        // Añadimos al costo total
        total_cost += tiempo_viaje;
    }

    // Guardamos el fitness en la población
    poblacion[idx].fitness = total_cost;
}

__global__ void seleccionar_padres_kernel(individuo_gpu *poblacion, individuo_gpu *padres,
                                          int num_competidores, int tamano_poblacion,
                                          int longitud_genotipo, hiprandState *states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tamano_poblacion) return;

    int mejor_idx = -1;
    double mejor_fitness = 1e9;

    for (int i = 0; i < num_competidores; i++) {
        int rand_idx = hiprand(&states[idx]) % tamano_poblacion;
        if (poblacion[rand_idx].fitness < mejor_fitness) {
            mejor_fitness = poblacion[rand_idx].fitness;
            mejor_idx = rand_idx;
        }
    }

    // Copiar el mejor individuo al arreglo de padres
    for (int j = 0; j < longitud_genotipo; j++) {
        padres[idx].genotipo[j] = poblacion[mejor_idx].genotipo[j];
    }
    padres[idx].fitness = poblacion[mejor_idx].fitness;
}

__global__ void cruzar_individuos_kernel(individuo_gpu *padres, individuo_gpu *hijos,
                                         double *distancias, double *ventanas_de_tiempo, double prob_cruce,
                                         int tamano_poblacion, int longitud_genotipo,
                                         int m, hiprandState *states)
{
    // 1) Cada bloque tiene "blockDim.x" hilos. 
    //    Usamos "extern __shared__ int sMem[]" para la memoria compartida dinámica.
    extern __shared__ unsigned char sMem[]; 


    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tamano_poblacion / 2) return;

    // blockSize = blockDim.x
    // Cada hilo "i" dentro del bloque usará un trozo de sMem

    size_t espacioCrossover = 3UL * longitud_genotipo * sizeof(int);  // hijo1,hijo2,visitado
    size_t espacioHeurRuta  = (size_t)longitud_genotipo * sizeof(int);
    size_t espacioHeurDist  = (size_t)longitud_genotipo * sizeof(DistanciaOrdenadaGPU);

    size_t totalPorHilo = espacioCrossover + espacioHeurRuta + espacioHeurDist;

    // offset para ESTE hilo (threadIdx.x)
    size_t offset = (size_t)threadIdx.x * totalPorHilo; 
    unsigned char* ptrBase = &sMem[offset];

    // 1) Crossover
    // a) hijo1: int[longitud_genotipo]
    int* hijo1 = reinterpret_cast<int*>(ptrBase);
    ptrBase += longitud_genotipo * sizeof(int);

    // b) hijo2: int[longitud_genotipo]
    int* hijo2 = reinterpret_cast<int*>(ptrBase);
    ptrBase += longitud_genotipo * sizeof(int);

    // c) visitado: int[longitud_genotipo]
    int* visitado = reinterpret_cast<int*>(ptrBase);
    ptrBase += longitud_genotipo * sizeof(int);

    // 2) Heurística
    // a) ruta_temp: int[longitud_genotipo]
    int* ruta_temp = reinterpret_cast<int*>(ptrBase);
    ptrBase += longitud_genotipo * sizeof(int);

    // b) dist_ordenadas: DistanciaOrdenadaGPU[longitud_genotipo]
    DistanciaOrdenadaGPU* dist_ordenadas = reinterpret_cast<DistanciaOrdenadaGPU*>(ptrBase);
    ptrBase += longitud_genotipo * sizeof(DistanciaOrdenadaGPU);

    // idx2 indica qué par (padre1, padre2) estamos trabajando
    int idx2 = idx * 2;

    // Decidimos si hacemos cruce
    if (hiprand_uniform(&states[idx]) < prob_cruce)
    {
        // 1) Generar hijo1 con cycle_crossover_device(padre1, padre2)
        cycle_crossover_device(
            padres[idx2].genotipo,
            padres[idx2 + 1].genotipo,
            hijo1,         // => en shared memory
            visitado,      // => en shared memory
            longitud_genotipo
        );

        // 2) Generar hijo2 con cycle_crossover_device(padre2, padre1)
        //    *Pero* hay que "reiniciar" "visitado" antes de reusar. Lo más fácil:
        //    reusar la misma "visitado[]" si deseas, o usar un trozo distinto.
        //    Aquí, por simplicidad, volvemos a poner en 0:
        for (int i = 0; i < longitud_genotipo; i++) {
            visitado[i] = 0;
        }
        cycle_crossover_device(
            padres[idx2 + 1].genotipo,
            padres[idx2].genotipo,
            hijo2,
            visitado,
            longitud_genotipo
        );

        // (Opcional) Llamar heurística:
        heuristica_abruptos_gpu(hijo1, longitud_genotipo, m, distancias, ventanas_de_tiempo,ruta_temp, dist_ordenadas);

        heuristica_abruptos_gpu(hijo2, longitud_genotipo, m, distancias, ventanas_de_tiempo,ruta_temp, dist_ordenadas);

        // 3) Evaluar padres e hijos
        double fit_p1 = evaluar_individuo_gpu(padres[idx2].genotipo, distancias, ventanas_de_tiempo, longitud_genotipo);
        double fit_p2 = evaluar_individuo_gpu(padres[idx2+1].genotipo, distancias, ventanas_de_tiempo, longitud_genotipo);
        double fit_h1 = evaluar_individuo_gpu(hijo1, distancias, ventanas_de_tiempo, longitud_genotipo);
        double fit_h2 = evaluar_individuo_gpu(hijo2, distancias, ventanas_de_tiempo, longitud_genotipo);

        // Seleccionamos 2 mejores
        double fitness_array[4] = { fit_p1, fit_p2, fit_h1, fit_h2 };
        int *genotipos[4]       = { padres[idx2].genotipo,
                                    padres[idx2+1].genotipo,
                                    hijo1,
                                    hijo2 };

        int mejores[2] = {0, 1};
        for (int j = 2; j < 4; j++) {
            if (fitness_array[j] < fitness_array[mejores[0]]) {
                mejores[1] = mejores[0];
                mejores[0] = j;
            }
            else if (fitness_array[j] < fitness_array[mejores[1]]) {
                mejores[1] = j;
            }
        }

        // Copiar a hijos finales
        for(int j = 0; j < longitud_genotipo; j++) {
            hijos[idx2].genotipo[j]   = genotipos[mejores[0]][j];
            hijos[idx2+1].genotipo[j] = genotipos[mejores[1]][j];
        }
        hijos[idx2].fitness   = fitness_array[mejores[0]];
        hijos[idx2+1].fitness = fitness_array[mejores[1]];
    }
    else
    {
        // Sin cruce, copiamos padres
        for (int i = 0; i < longitud_genotipo; i++) {
            hijos[idx2].genotipo[i]   = padres[idx2].genotipo[i];
            hijos[idx2+1].genotipo[i] = padres[idx2+1].genotipo[i];
        }
        hijos[idx2].fitness   = padres[idx2].fitness;
        hijos[idx2+1].fitness = padres[idx2+1].fitness;
    }
}

__global__ void mutar_individuos_kernel(individuo_gpu *individuos, double *distancias, double *ventanas_de_tiempo,
                                        double prob_mutacion, int tamano_poblacion,
                                        int longitud_genotipo, hiprandState *states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tamano_poblacion) return;

    if (hiprand_uniform(&states[idx]) < prob_mutacion) {
        int idx1 = (int)(hiprand_uniform(&states[idx]) * longitud_genotipo);
        int idx2 = (int)(hiprand_uniform(&states[idx]) * longitud_genotipo);
        while (idx2 == idx1) {
            idx2 = (int)(hiprand_uniform(&states[idx]) * longitud_genotipo);
        }
        int temp = individuos[idx].genotipo[idx1];
        individuos[idx].genotipo[idx1] = individuos[idx].genotipo[idx2];
        individuos[idx].genotipo[idx2] = temp;

        // recalcular fitness local
        double total_cost = 0.0;
        for (int i = 0; i < longitud_genotipo - 1; i++) {
            total_cost += distancias[individuos[idx].genotipo[i] * longitud_genotipo +
                                     individuos[idx].genotipo[i+1]];
        }
        total_cost += distancias[individuos[idx].genotipo[longitud_genotipo - 1] * 
                                 longitud_genotipo + individuos[idx].genotipo[0]];
        individuos[idx].fitness = total_cost;
    }
}

// ----------------------------------------------------
// Funciones device auxiliares
// ----------------------------------------------------

/// --------------------------------------------------------------------------
/// cycle_crossover_device:
///   Versión "device" de tu ciclo de cruce en CPU.
///   Genera 1 hijo en el array "child" (size = num_ciudades)
///   usando padre1 "p1" y padre2 "p2" (también arrays de size=num_ciudades).
/// --------------------------------------------------------------------------

// ---------------------------------------------------------------------
// cycle_crossover_device: Sin "new[]".
// Usa "child[]" y "visitado[]" que fueron asignados en shared memory.
// ---------------------------------------------------------------------
__device__ void cycle_crossover_device(const int *p1, const int *p2,
                                       int *child, int *visitado,
                                       int num_ciudades)
{
    // 1) Inicializa child con -1, visitado = 0
    for (int i = 0; i < num_ciudades; i++) {
        child[i]   = -1;
        visitado[i] = 0;  // 0 => no visitado
    }

    int ciclo = 0;
    int restantes = num_ciudades;

    // 2) Mientras queden posiciones sin visitar
    while (restantes > 0) {
        // encontrar primera posición no visitada
        int inicio = -1;
        for (int i = 0; i < num_ciudades; i++) {
            if (visitado[i] == 0) {
                inicio = i;
                break;
            }
        }

        ciclo++;
        int actual = inicio;

        // 3) Recorremos el ciclo
        while (true) {
            visitado[actual] = 1;
            restantes--;

            // En ciclos impares copiamos de p1, en pares de p2
            if (ciclo % 2 == 1) {
                child[actual] = p1[actual];
            } else {
                child[actual] = p2[actual];
            }

            // "valor_buscar" = p2[actual], lo buscamos en p1
            int valor_buscar = p2[actual];
            int siguiente = -1;
            for (int j = 0; j < num_ciudades; j++) {
                if (p1[j] == valor_buscar) {
                    siguiente = j;
                    break;
                }
            }
            if (siguiente == -1 || visitado[siguiente] == 1) {
                break;
            }
            actual = siguiente;
        }
    }
}

__device__ double evaluar_individuo_gpu(int *ruta, double *distancias, double *ventanas_de_tiempo, int num_ciudades) {
    double total_cost = 0.0;      // Costo total del recorrido (en horas)
    double tiempo_acumulado = 0.0; // Tiempo transcurrido desde el inicio del recorrido

    // Iteramos sobre las ciudades en la ruta de forma circular
    for (int i = 0; i < num_ciudades; i++) {
        int origen = ruta[i];
        int destino = ruta[(i + 1) % num_ciudades];

        // Añadimos el tiempo de viaje entre ciudades
        double tiempo_viaje = distancias[origen * num_ciudades + destino];
        tiempo_acumulado += tiempo_viaje;

        // Calculamos la hora de llegada al destino (ajustado al formato de 24 horas)
        double hora_llegada = fmod(tiempo_acumulado, 24.0);

        // Ventanas de tiempo de la ciudad de destino
        double ventana_inicio = ventanas_de_tiempo[destino * 2];
        double ventana_fin = ventanas_de_tiempo[destino * 2 + 1];

        // Ajustamos el tiempo acumulado si se llega fuera de la ventana permitida
        if (hora_llegada < ventana_inicio) {
            // Esperamos hasta el inicio de la ventana
            tiempo_acumulado += (ventana_inicio - hora_llegada);
        } else if (ventana_fin < ventana_inicio) {
            // Caso especial: La ventana cruza medianoche (ej. 22:00 a 02:00)
            if (hora_llegada > ventana_fin && hora_llegada < ventana_inicio) {
                tiempo_acumulado += (24.0 - hora_llegada + ventana_inicio); // Esperamos al siguiente día
            }
        } else if (hora_llegada > ventana_fin) {
            // Esperamos al siguiente día si llegamos después del cierre
            tiempo_acumulado += (24.0 - hora_llegada + ventana_inicio);
        }

        // Añadimos al costo total
        total_cost += tiempo_viaje;
    }

    return total_cost;
}

__device__ void heuristica_abruptos_gpu(int *ruta,
                                        int num_ciudades,
                                        int m,
                                        double *distancias,
                                        double *ventanas_de_tiempo,
                                        int *ruta_temp,
                                        DistanciaOrdenadaGPU *dist_ordenadas) {

    for (int i = 0; i < num_ciudades; i++) {
        int ciudad_actual = ruta[i];
        
        // Ordenar ciudades por distancia
        for (int j = 0; j < num_ciudades; j++) {
            dist_ordenadas[j].distancia = distancias[ciudad_actual * num_ciudades + j];
            dist_ordenadas[j].indice = j;
        }
        
        // Ordenamiento simple para GPU
        for (int j = 0; j < m; j++) {
            for (int k = j + 1; k < num_ciudades; k++) {
                if (comparar_distancias_gpu(dist_ordenadas[k], dist_ordenadas[j])) {
                    DistanciaOrdenadaGPU temp = dist_ordenadas[j];
                    dist_ordenadas[j] = dist_ordenadas[k];
                    dist_ordenadas[k] = temp;
                }
            }
        }

        int pos_actual = -1;
        for (int j = 0; j < num_ciudades; j++) {
            if (ruta[j] == ciudad_actual) {
                pos_actual = j;
                break;
            }
        }

        double mejor_costo = evaluar_individuo_gpu(ruta, distancias, ventanas_de_tiempo, num_ciudades);
        int mejor_posicion = pos_actual;
        int mejor_vecino = -1;

        for (int j = 1; j <= m && j < num_ciudades; j++) {
            int ciudad_cercana = dist_ordenadas[j].indice;
            
            int pos_cercana = -1;
            for (int k = 0; k < num_ciudades; k++) {
                if (ruta[k] == ciudad_cercana) {
                    pos_cercana = k;
                    break;
                }
            }

            if (pos_cercana != -1) {
                for (int posicion_antes_o_despues = 0; posicion_antes_o_despues <= 1; posicion_antes_o_despues++) {
                    // Copiar ruta actual
                    for (int k = 0; k < num_ciudades; k++) {
                        ruta_temp[k] = ruta[k];
                    }
                    
                    eliminar_de_posicion_gpu(ruta_temp, num_ciudades, pos_actual);
                    
                    int nueva_pos = pos_cercana + posicion_antes_o_despues;
                    if (nueva_pos > pos_actual) nueva_pos--;
                    if (nueva_pos >= num_ciudades) nueva_pos = num_ciudades - 1;
                    
                    insertar_en_posicion_gpu(ruta_temp, num_ciudades, ciudad_actual, nueva_pos);
                    
                    double nuevo_costo = evaluar_individuo_gpu(ruta_temp, distancias, ventanas_de_tiempo, num_ciudades);
                    
                    if (nuevo_costo < mejor_costo) {
                        mejor_costo = nuevo_costo;
                        mejor_posicion = nueva_pos;
                        mejor_vecino = ciudad_cercana;
                    }
                }
            }
        }

        if (mejor_vecino != -1 && mejor_posicion != pos_actual) {
            for (int k = 0; k < num_ciudades; k++) {
                ruta_temp[k] = ruta[k];
            }
            eliminar_de_posicion_gpu(ruta_temp, num_ciudades, pos_actual);
            insertar_en_posicion_gpu(ruta_temp, num_ciudades, ciudad_actual, mejor_posicion);
            for (int k = 0; k < num_ciudades; k++) {
                ruta[k] = ruta_temp[k];
            }
        }
    }

}

__device__ int comparar_distancias_gpu(DistanciaOrdenadaGPU a, DistanciaOrdenadaGPU b) {
    return (a.distancia < b.distancia);
}

__device__ void insertar_en_posicion_gpu(int* array, int longitud, int elemento, int posicion) {
    for (int i = longitud-1; i > posicion; i--) {
        array[i] = array[i-1];
    }
    array[posicion] = elemento;
}

__device__ void eliminar_de_posicion_gpu(int* array, int longitud, int posicion) {
    int valor = array[posicion];
    for (int i = posicion; i < longitud-1; i++) {
        array[i] = array[i+1];
    }
    array[longitud-1] = valor;
}

// ----------------------------------------------------
// Funciones para copiar poblaciones CPU <-> GPU
// ----------------------------------------------------
void copiarPoblacionCPUaGPU(const poblacion *pobCPU, 
                            individuo_gpu *pobGPU, 
                            int *genotiposGPU,
                            int tamPobl, 
                            int longGen)
{
    // Array temporal en CPU
    individuo_gpu *temp = (individuo_gpu*)malloc(tamPobl * sizeof(individuo_gpu));

    for(int i = 0; i < tamPobl; i++) {
        // Copiar genotipo
        gpuErrchk(hipMemcpy(genotiposGPU + i*longGen,
                             pobCPU->individuos[i].genotipo,
                             longGen*sizeof(int),
                             hipMemcpyHostToDevice));
        // Ajustamos puntero
        temp[i].genotipo = genotiposGPU + i*longGen;
        // Fitness
        temp[i].fitness = pobCPU->individuos[i].fitness;
    }

    // Copiar a pobGPU
    gpuErrchk(hipMemcpy(pobGPU,
                         temp,
                         tamPobl*sizeof(individuo_gpu),
                         hipMemcpyHostToDevice));

    free(temp);
}

void copiarPoblacionGPUaCPU(poblacion *pobCPU, 
                            const individuo_gpu *pobGPU, 
                            const int *genotiposGPU,
                            int tamPobl, 
                            int longGen)
{
    individuo_gpu *temp = (individuo_gpu*)malloc(tamPobl * sizeof(individuo_gpu));

    gpuErrchk(hipMemcpy(temp,
                         pobGPU,
                         tamPobl*sizeof(individuo_gpu),
                         hipMemcpyDeviceToHost));

    for(int i = 0; i < tamPobl; i++) {
        gpuErrchk(hipMemcpy(pobCPU->individuos[i].genotipo,
                             genotiposGPU + i*longGen,
                             longGen*sizeof(int),
                             hipMemcpyDeviceToHost));
        pobCPU->individuos[i].fitness = temp[i].fitness;
    }
    free(temp);
}

// ----------------------------------------------------
// Funciones para crear y manejar poblaciones en CPU
// ----------------------------------------------------
poblacion *crear_poblacion(int tamano, int longitud_genotipo) {
    poblacion *Poblacion = (poblacion *)malloc(sizeof(poblacion));
    if(!Poblacion) {
        fprintf(stderr, "Error al asignar memoria para Poblacion\n");
        exit(EXIT_FAILURE);
    }
    Poblacion->tamano = tamano;
    Poblacion->individuos = (individuo *)malloc(tamano * sizeof(individuo));
    if(!Poblacion->individuos) {
        fprintf(stderr, "Error al asignar memoria para individuos\n");
        free(Poblacion);
        exit(EXIT_FAILURE);
    }
    for(int i=0; i<tamano; i++) {
        Poblacion->individuos[i].genotipo = (int*)malloc(longitud_genotipo*sizeof(int));
        if(!Poblacion->individuos[i].genotipo) {
            fprintf(stderr, "Error al asignar memoria para genotipo\n");
            for(int j=0; j<i; j++) {
                free(Poblacion->individuos[j].genotipo);
            }
            free(Poblacion->individuos);
            free(Poblacion);
            exit(EXIT_FAILURE);
        }
    }
    return Poblacion;
}

void crear_permutaciones(poblacion *poblacion, int longitud_genotipo) {
    for(int i=0; i< poblacion->tamano; i++) {
        // inicializa
        for(int j=0; j<longitud_genotipo; j++) {
            poblacion->individuos[i].genotipo[j] = j;
        }
        // fisher-yates
        for(int j = longitud_genotipo-1; j>0; j--) {
            int k = rand()%(j+1);
            int tmp = poblacion->individuos[i].genotipo[j];
            poblacion->individuos[i].genotipo[j] = poblacion->individuos[i].genotipo[k];
            poblacion->individuos[i].genotipo[k] = tmp;
        }
    }
}

void ordenar_poblacion(poblacion *poblacion) {
    int n = poblacion->tamano;
    if(n<=1) return;
    int profundidad_max = 2 * log2_suelo(n);
    introsort_util(poblacion->individuos, &profundidad_max, 0, n);
}

void actualizar_poblacion(poblacion **destino, poblacion *origen, int longitud_genotipo) {
    poblacion *nueva = crear_poblacion(origen->tamano, longitud_genotipo);
    for(int i=0; i<origen->tamano; i++) {
        for(int j=0; j<longitud_genotipo; j++) {
            nueva->individuos[i].genotipo[j] = origen->individuos[i].genotipo[j];
        }
        nueva->individuos[i].fitness = origen->individuos[i].fitness;
    }
    if(*destino!=NULL) {
        for(int i=0; i<(*destino)->tamano; i++) {
            free((*destino)->individuos[i].genotipo);
        }
        free((*destino)->individuos);
        free(*destino);
    }
    *destino = nueva;
}

void liberar_poblacion(poblacion *pob) {
    if(!pob) return;
    if(pob->individuos) {
        for(int i=0; i<pob->tamano; i++) {
            free(pob->individuos[i].genotipo);
        }
        free(pob->individuos);
    }
    free(pob);
}

// ----------------------------------------------------
// Funciones de ordenamiento (introsort, etc.)
// ----------------------------------------------------
int log2_suelo(int n) {
    int log = 0;
    while(n>1) {
        n >>= 1;
        log++;
    }
    return log;
}

void introsort_util(individuo *arr, int *profundidad_max, int inicio, int fin) {
    int tamano = fin - inicio;
    if(tamano<16) {
        insertion_sort(arr, inicio, fin-1);
        return;
    }
    if(*profundidad_max == 0) {
        heapsort(arr+inicio, tamano);
        return;
    }
    (*profundidad_max)--;
    int piv = particion(arr, inicio, fin-1);
    introsort_util(arr, profundidad_max, inicio,   piv);
    introsort_util(arr, profundidad_max, piv+1,    fin);
}

int particion(individuo *arr, int bajo, int alto) {
    int medio = bajo + (alto-bajo)/2;
    int indice_pivote = mediana_de_tres(arr, bajo, medio, alto);
    intercambiar_individuos(&arr[indice_pivote], &arr[alto]);

    individuo pivote = arr[alto];
    int i = bajo-1;
    for(int j=bajo; j<alto; j++) {
        if(arr[j].fitness <= pivote.fitness) {
            i++;
            intercambiar_individuos(&arr[i], &arr[j]);
        }
    }
    intercambiar_individuos(&arr[i+1], &arr[alto]);
    return i+1;
}

int mediana_de_tres(individuo *arr, int a, int b, int c) {
    if(arr[a].fitness <= arr[b].fitness) {
        if(arr[b].fitness <= arr[c].fitness) return b;
        else if(arr[a].fitness <= arr[c].fitness) return c;
        else return a;
    } else {
        if(arr[a].fitness <= arr[c].fitness) return a;
        else if(arr[b].fitness <= arr[c].fitness) return c;
        else return b;
    }
}

void intercambiar_individuos(individuo *a, individuo *b) {
    individuo temp = *a;
    *a = *b;
    *b = temp;
}

void insertion_sort(individuo *arr, int izquierda, int derecha) {
    for(int i=izquierda+1; i<=derecha; i++) {
        individuo clave = arr[i];
        int j = i-1;
        while(j>=izquierda && arr[j].fitness > clave.fitness) {
            arr[j+1] = arr[j];
            j--;
        }
        arr[j+1] = clave;
    }
}

void heapsort(individuo *arr, int n) {
    for(int i = n/2 -1; i>=0; i--)
        heapify(arr, n, i);
    for(int i=n-1; i>0; i--) {
        intercambiar_individuos(&arr[0], &arr[i]);
        heapify(arr, i, 0);
    }
}

void heapify(individuo *arr, int n, int i) {
    int mayor = i;
    int izquierda = 2*i + 1;
    int derecha   = 2*i + 2;
    if(izquierda<n && arr[izquierda].fitness > arr[mayor].fitness) {
        mayor = izquierda;
    }
    if(derecha<n && arr[derecha].fitness > arr[mayor].fitness) {
        mayor = derecha;
    }
    if(mayor!=i) {
        intercambiar_individuos(&arr[i], &arr[mayor]);
        heapify(arr, n, mayor);
    }
}
